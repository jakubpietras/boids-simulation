#include "hip/hip_runtime.h"
#include "simulationGPU.cuh"
#include <glad/glad.h>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <cuda_gl_interop.h>
#include <>
#include <glm/ext/matrix_transform.hpp>
#include "float4x4.h"
#include <hip/hip_math_constants.h>
#include <cmath>

#define THREAD_COUNT 256

__device__ 
float3 operator+(const float3& a, const float3& b) 
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}
__device__ 
float3 operator-(const float3& a, const float3& b) 
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ 
float3 operator*(const float3& a, float b) 
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ 
float3 operator/(const float3& a, float b) 
{
    float inv = 1.0f / b;
    return a * inv;
}

__device__ 
float length(const float3& v) 
{
    return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ 
float3 normalize(const float3& v) 
{
    float len = length(v);
    if (len > 0) return v / len;
    return v;
}

__device__ 
float dot(const float3& a, const float3& b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__global__ void computeFlockingForcesKernel(
    float3* positions, 
    float3* velocities, 
    float3* forces,
    int* cellStart, 
    int* cellEnd, 
    int* boidIndices,
    int totalBoids, 
    int gridSize, 
    float cellSize,
    float visionRadius, 
    float visionAngle, 
    float maxSpeed,
    float alignmentFactor, 
    float cohesionFactor, 
    float separationFactor)
{
    int boidIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (boidIdx >= totalBoids) return;

    float3 pos = positions[boidIdx];
    float3 vel = velocities[boidIdx];

    float3 avgVelocity = make_float3(0, 0, 0);
    float3 avgPosition = make_float3(0, 0, 0);
    float3 separation = make_float3(0, 0, 0);
    int neighborCount = 0;

    int cellX = static_cast<int>(pos.x / cellSize);
    int cellY = static_cast<int>(pos.y / cellSize);
    int cellZ = static_cast<int>(pos.z / cellSize);

    const float MIN_DISTANCE = 2.0f;

    // loop over 3x3x3 grid around the boid's cell
    for (int dz = -1; dz <= 1; dz++) 
    {
        for (int dy = -1; dy <= 1; dy++) 
        {
            for (int dx = -1; dx <= 1; dx++) 
            {
                int nx = cellX + dx;
                int ny = cellY + dy;
                int nz = cellZ + dz;

                if (nx < 0 || nx >= gridSize ||
                    ny < 0 || ny >= gridSize ||
                    nz < 0 || nz >= gridSize)
                    continue;

                int cellIdx = nx + ny * gridSize + nz * gridSize * gridSize;
                int start = cellStart[cellIdx]; // where the indices of boids in that cell start
                int end = cellEnd[cellIdx]; // where the indices of boids in that cell end

                if (start == -1 || end == -1) continue;

                for (int otherIdx = start; otherIdx <= end; otherIdx++) 
                {
                    int idx = boidIndices[otherIdx];
                    if (idx != boidIdx) 
                    {
                        float3 otherPos = positions[idx];
                        float3 diff = pos - otherPos;
                        float dist = length(diff);

                        if (dist < visionRadius && dist > 0) 
                        {
                            float3 otherVel = velocities[idx];

                            avgVelocity = avgVelocity + otherVel;
                            avgPosition = avgPosition + otherPos;

                            float separationStrength = (dist < MIN_DISTANCE) ?
                                5.0f * (MIN_DISTANCE - dist) / MIN_DISTANCE : 1.0f; // stronger separation when too close
                            separation = separation + normalize(diff) * separationStrength / dist;
                            neighborCount++;
                        }
                    }
                }
            }
        }
    }

    float3 force = make_float3(0, 0, 0);
    if (neighborCount > 0) 
    {
        avgVelocity = avgVelocity / neighborCount;
        float3 alignmentForce = (normalize(avgVelocity) * maxSpeed - vel) * alignmentFactor;

        avgPosition = avgPosition / neighborCount;
        float3 cohesionForce = (normalize(avgPosition - pos) * maxSpeed - vel) * cohesionFactor;

        separation = separation / neighborCount;
        float3 separationForce = (normalize(separation) * maxSpeed - vel) * separationFactor;

        force = alignmentForce + cohesionForce + separationForce;

        // limit the speed
        float forceMag = length(force);
        if (forceMag > maxSpeed)
            force = normalize(force) * maxSpeed;
    }
    forces[boidIdx] = force;
}

__global__ void updatePositionsKernel(
    float3* positions, 
    float3* velocities, 
    float3* forces,
    int totalBoids, 
    float deltaTime, 
    float maxSpeed,
    int boxSize, 
    float wallMargin)
{
    int boidIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (boidIdx >= totalBoids) return;

    float3 pos = positions[boidIdx];
    float3 vel = velocities[boidIdx];
    float3 force = forces[boidIdx];

    vel = vel + force * deltaTime;
    float3 steeringForce = make_float3(0.0f, 0.0f, 0.0f);

    if (pos.x < wallMargin) 
    {
        steeringForce.x += (wallMargin - pos.x) / wallMargin * maxSpeed;
        if (pos.x <= 0) 
        {
            pos.x = 0;
            vel.x = abs(vel.x);
        }
    }
    else if (pos.x > boxSize - wallMargin) 
    {
        steeringForce.x -= (pos.x - (boxSize - wallMargin)) / wallMargin * maxSpeed;
        if (pos.x >= boxSize) 
        {
            pos.x = boxSize;
            vel.x = -abs(vel.x);
        }
    }

    if (pos.y < wallMargin) 
    {
        steeringForce.y += (wallMargin - pos.y) / wallMargin * maxSpeed;
        if (pos.y <= 0) 
        {
            pos.y = 0;
            vel.y = abs(vel.y);
        }
    }
    else if (pos.y > boxSize - wallMargin) 
    {
        steeringForce.y -= (pos.y - (boxSize - wallMargin)) / wallMargin * maxSpeed;
        if (pos.y >= boxSize) 
        {
            pos.y = boxSize;
            vel.y = -abs(vel.y);
        }
    }

    if (pos.z < wallMargin) 
    {
        steeringForce.z += (wallMargin - pos.z) / wallMargin * maxSpeed;
        if (pos.z <= 0) 
        {
            pos.z = 0;
            vel.z = abs(vel.z);
        }
    }
    else if (pos.z > boxSize - wallMargin) 
    {
        steeringForce.z -= (pos.z - (boxSize - wallMargin)) / wallMargin * maxSpeed;
        if (pos.z >= boxSize) 
        {
            pos.z = boxSize;
            vel.z = -abs(vel.z);
        }
    }

    float steeringMagnitude = length(steeringForce);
    if (steeringMagnitude > maxSpeed) 
    {
        steeringForce = normalize(steeringForce) * maxSpeed;
    }

    vel = vel + steeringForce * deltaTime;

    float speed = length(vel);
    if (speed > maxSpeed) 
    {
        vel = normalize(vel) * maxSpeed;
    }

    pos = pos + vel * deltaTime;
    positions[boidIdx] = pos;
    velocities[boidIdx] = vel;
}

__global__
void updateModelMatricesKernel(float4x4* matrices, float3* position, float3* velocity, int totalBoids)
{
    float scale = 1.0f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= totalBoids)
        return;

    glm::vec3 vel = { velocity[idx].x, velocity[idx].y, velocity[idx].z };
    glm::vec3 pos = { position[idx].x, position[idx].y, position[idx].z };
    glm::vec3 scaleVec = glm::vec3(scale);
    glm::mat4 model = glm::mat4(1.0f);
    glm::vec3 modelOrientation = { 1.0f, 0.0f, 0.0f }; // orientation of the base model
    glm::vec3 velocityNormalized = glm::normalize(vel);
    glm::vec3 rotationAxis = glm::cross(modelOrientation, velocityNormalized);
    float rotationAngle = glm::acos(glm::dot(modelOrientation, velocityNormalized));

    model = glm::translate(model, pos);
    model = glm::scale(model, scaleVec);
    if (rotationAxis != glm::vec3(0.0f))
        model = glm::rotate(model, rotationAngle, glm::normalize(rotationAxis));

    float4x4 result;
    result.col0 = make_float4(model[0][0], model[0][1], model[0][2], model[0][3]);
    result.col1 = make_float4(model[1][0], model[1][1], model[1][2], model[1][3]);
    result.col2 = make_float4(model[2][0], model[2][1], model[2][2], model[2][3]);
    result.col3 = make_float4(model[3][0], model[3][1], model[3][2], model[3][3]);

    matrices[idx] = result;
}

SimulationGPU& SimulationGPU::getInstance() 
{
    static SimulationGPU instance;
    return instance;
}

void SimulationGPU::init(std::shared_ptr<Boids> boidsPtr, int boxSize, unsigned int modelMatVBO)
{
    vbo = modelMatVBO;
    hipError_t err = hipGraphicsGLRegisterBuffer(&vboRes, vbo, cudaGraphicsMapFlagsNone);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to register VBO with CUDA: " << hipGetErrorString(err) << std::endl;
        throw std::runtime_error("hipGraphicsGLRegisterBuffer failed");
    }

    err = hipMalloc(&d_positions, boidsPtr->totalBoids * sizeof(float3));
    if (err != hipSuccess) 
        throw std::runtime_error("Failed to allocate d_positions");

    err = hipMalloc(&d_velocities, boidsPtr->totalBoids * sizeof(float3));
    if (err != hipSuccess) 
        throw std::runtime_error("Failed to allocate d_velocities");

    err = hipMalloc(&d_forces, boidsPtr->totalBoids * sizeof(float3));
    if (err != hipSuccess) 
        throw std::runtime_error("Failed to allocate d_forces");

    err = hipMemcpy(d_positions, boidsPtr->h_position.data(),
        boidsPtr->totalBoids * sizeof(float3), hipMemcpyHostToDevice);
    if (err != hipSuccess) 
        throw std::runtime_error("Failed to copy positions");

    err = hipMemcpy(d_velocities, boidsPtr->h_velocity.data(),
        boidsPtr->totalBoids * sizeof(float3), hipMemcpyHostToDevice);
    if (err != hipSuccess) 
        throw std::runtime_error("Failed to copy velocities");

    float cellSize = 5.0f;
    boids = boidsPtr;
    grid = std::make_shared<SpatialGridGPU>(cellSize, boxSize, boids->totalBoids);
}

SimulationGPU::~SimulationGPU() 
{
    hipError_t err = hipGraphicsUnregisterResource(vboRes);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to unregister VBO with CUDA: " << hipGetErrorString(err) << std::endl;
    }
    hipFree(d_positions);
    hipFree(d_velocities);
    hipFree(d_forces);
}

void SimulationGPU::run(const SimulationState& state, float deltaTime) 
{
    hipError_t err;
    if (!state.isPlaying) return;

    err = hipGraphicsMapResources(1, &vboRes, 0);
    if (err != hipSuccess) 
    {
        std::cerr << "ERROR: hipGraphicsMapResources failed: " << hipGetErrorString(err) << std::endl;
        return;
    }

    size_t numBytes = boids->totalBoids * sizeof(float4x4);
    float4x4* modelMatrices;
    err = hipGraphicsResourceGetMappedPointer((void**)&modelMatrices, &numBytes, vboRes);
    if (err != hipSuccess) 
    {
        std::cerr << "ERROR: hipGraphicsResourceGetMappedPointer failed: " << hipGetErrorString(err) << std::endl;
        hipGraphicsUnmapResources(1, &vboRes, 0);
        return;
    }

    const float wallMargin = 1.0f;
    int numBlocks = std::ceil(static_cast<float>(boids->totalBoids) / THREAD_COUNT);

    grid->update(d_positions, boids->totalBoids);
    int* d_cellStart;
    int* d_cellEnd;
    int* d_boidIndices;
    grid->getGridData(d_cellStart, d_cellEnd, d_boidIndices);

    computeFlockingForcesKernel << <numBlocks, THREAD_COUNT >> > (
        d_positions,
        d_velocities,
        d_forces,
        d_cellStart,
        d_cellEnd,
        d_boidIndices,
        boids->totalBoids,
        static_cast<int>(ceil(state.boxSize / 5.0f)),
        5.0f,
        state.visionRadius,
        state.visionAngle,
        state.maxSpeed,
        state.alignmentFactor,
        state.cohesionFactor,
        state.separationFactor
        );

    updatePositionsKernel << <numBlocks, THREAD_COUNT >> > (
        d_positions,
        d_velocities,
        d_forces,
        boids->totalBoids,
        deltaTime,
        state.maxSpeed,
        state.boxSize,
        wallMargin
        );

    updateModelMatricesKernel << <numBlocks, THREAD_COUNT >> > (
        modelMatrices,
        d_positions,
        d_velocities,
        boids->totalBoids
        );

    err = hipGetLastError();
    if (err != hipSuccess) 
    {
        std::cerr << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipGraphicsUnmapResources(1, &vboRes, 0);
        return;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) 
    {
        std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
        hipGraphicsUnmapResources(1, &vboRes, 0);
        return;
    }

    err = hipGraphicsUnmapResources(1, &vboRes, 0);
    if (err != hipSuccess) 
    {
        std::cerr << "ERROR: hipGraphicsUnmapResources failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
}

void SimulationGPU::reset()
{
    hipError_t err;

    // unregister the VBO resource
    if (vboRes)
    {
        err = hipGraphicsUnregisterResource(vboRes);
        if (err != hipSuccess)
        {
            std::cerr << "ERROR: hipGraphicsUnregisterResource failed during reset: " << hipGetErrorString(err) << std::endl;
        }
        vboRes = nullptr;
    }

    // re-register the VBO resource with CUDA
    err = hipGraphicsGLRegisterBuffer(&vboRes, vbo, cudaGraphicsMapFlagsNone);
    if (err != hipSuccess)
    {
        std::cerr << "ERROR: hipGraphicsGLRegisterBuffer failed during reset: " << hipGetErrorString(err) << std::endl;
        throw std::runtime_error("hipGraphicsGLRegisterBuffer failed");
    }
}
