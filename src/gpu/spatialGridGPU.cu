#include "hip/hip_runtime.h"
#include "spatialGridGPU.cuh"

#include <hip/hip_runtime.h>
#include <>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#define THREAD_COUNT 256

__global__ void assignBoidsToCellsKernel(float3* positions, int* cellIndices, int* boidIndices,
    float cellSize, int gridSize, int totalBoids) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= totalBoids) return;

    float3 pos = positions[idx];
    int cellX = min(max(static_cast<int>(pos.x / cellSize), 0), gridSize - 1);
    int cellY = min(max(static_cast<int>(pos.y / cellSize), 0), gridSize - 1);
    int cellZ = min(max(static_cast<int>(pos.z / cellSize), 0), gridSize - 1);

    cellIndices[idx] = cellX + cellY * gridSize + cellZ * gridSize * gridSize;
    boidIndices[idx] = idx;
}

__global__ void computeCellRangesKernel(int* cellIndices,
    int* cellStart, int* cellEnd, int totalBoids) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= totalBoids) return;

    int cellIdx = cellIndices[idx];
    if (idx == 0 || cellIndices[idx - 1] != cellIdx) 
    {
        cellStart[cellIdx] = idx;
    }
    if (idx == totalBoids - 1 || cellIndices[idx + 1] != cellIdx) 
    {
        cellEnd[cellIdx] = idx;
    }
}

SpatialGridGPU::SpatialGridGPU(float cellSize, int boxSize, int totalBoids)
    : cellSize(cellSize) 
{
    gridSize = ceil(boxSize / cellSize);
    totalCells = gridSize * gridSize * gridSize;

    // Allocate device memory
    hipMalloc(&d_cellIndices, totalBoids * sizeof(int));
    hipMalloc(&d_boidIndices, totalBoids * sizeof(int));
    hipMalloc(&d_cellStart, totalCells * sizeof(int));
    hipMalloc(&d_cellEnd, totalCells * sizeof(int));
}

SpatialGridGPU::~SpatialGridGPU() 
{
    hipFree(d_cellIndices);
    hipFree(d_boidIndices);
    hipFree(d_cellStart);
    hipFree(d_cellEnd);
}

void SpatialGridGPU::update(float3* positions, int totalBoids) 
{
    int numBlocks = std::ceil(static_cast<float>(totalBoids) / THREAD_COUNT);

    hipMemset(d_cellStart, -1, totalCells * sizeof(int));
    hipMemset(d_cellEnd, -1, totalCells * sizeof(int));

    // Assign boids to cells
    assignBoidsToCellsKernel << <numBlocks, THREAD_COUNT >> > (
        positions, d_cellIndices, d_boidIndices, cellSize, gridSize, totalBoids);

    // Sort by cell indices
    thrust::device_ptr<int> cell_indices_ptr(d_cellIndices);
    thrust::device_ptr<int> boid_indices_ptr(d_boidIndices);
    thrust::sort_by_key(cell_indices_ptr, cell_indices_ptr + totalBoids, boid_indices_ptr);

    // Compute cell ranges
    computeCellRangesKernel << <numBlocks, THREAD_COUNT >> > (
        d_cellIndices, d_cellStart, d_cellEnd, totalBoids);
}

void SpatialGridGPU::getGridData(int*& d_cellStart, int*& d_cellEnd, int*& d_boidIndices) 
{
    d_cellStart = this->d_cellStart;
    d_cellEnd = this->d_cellEnd;
    d_boidIndices = this->d_boidIndices;
}